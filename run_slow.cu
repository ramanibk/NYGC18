#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#define NUCBITSZ 3
bool* getbool(char base){
  bool *n = new bool[3] {false, false, false};
  bool *a = new bool[3] {false, false, true };
  bool *t = new bool[3] {false, true, false };
  bool *g = new bool[3] {false, true, true  };
  bool *c = new bool[3] {true, false, false };
  bool *x = new bool[3] {true, true, true   };

  switch(base){
    case 'A': return a; break;
    case 'T': return t; break;
    case 'G': return g; break;
    case 'C': return c; break;
    case 'N': return n; break;
    case 'a': return a; break;
    case 't': return t; break;
    case 'g': return g; break;
    case 'c': return c; break;
    case 'n': return n; break;
    default : return x; break;
  }
}

__global__
void myKernel(bool* d_genome, bool* d_guides, bool* d_output, int GUIDESBATCHSZ){
  int startgenome = blockIdx.x * blockDim.x;
  int startwindow = startgenome + threadIdx.x;
   __shared__ bool blockGuide[20*3];
  int guidecnt = 0;
  while (guidecnt < (GUIDESBATCHSZ/3)/20 && startwindow < blockDim.x * gridDim.x ){
    for (int i = 0; i < 20*3; i++)
      blockGuide[i] = d_guides[20*3*guidecnt + i];
    bool resArray[20*3];
    for (int i = 0; i < 20*3; i++)
      resArray[i] = !(blockGuide[i] ^ d_genome[startwindow*3 + i]);
    bool res = true;
    for (int i = 0; i < 20*3; i++)
      res &= resArray[i];
    if (res) d_output[(gridDim.x * blockDim.x) * guidecnt + startwindow] = true;
    guidecnt += 1;
  }
}



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess){
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

int main(int argc, char **argv){

  FILE *fgenome, *fguides, *fexact, *fmis1, *fmis2;
  int GRIDSZ = 32, BLOCKSZ = 1024, GUIDESZ = 1024, GUIDELENGTH = 20, TOTGUIDES = 0, GENOMEBATCHSZ, GUIDESBATCHSZ, OUTPUTBATCHSZ;
  bool *h_genome, *h_guides, *h_output, *d_genome, *d_output, *d_guides;
  int inp, igenome, ibingen, ngenome = 0, nguides = 0, iguides, ibingui, absidxgui, absidxgen = 0;
  bool flaggenome = true, flagguides = true;
  int *exact, *mis1, *mis2;
 
  GENOMEBATCHSZ = (BLOCKSZ*GRIDSZ+GUIDELENGTH - 1)*NUCBITSZ;
  GUIDESBATCHSZ = GUIDESZ*GUIDELENGTH*NUCBITSZ;
  OUTPUTBATCHSZ = GUIDESZ*GRIDSZ*BLOCKSZ;

  hipFree(0);
  gpuErrchk(hipMalloc((void**)&d_genome, sizeof(bool)*GENOMEBATCHSZ));
  gpuErrchk(hipMalloc((void**)&d_guides, sizeof(bool)*GUIDESBATCHSZ));
  gpuErrchk(hipMalloc((void**)&d_output, sizeof(bool)*OUTPUTBATCHSZ));
  h_genome = (bool*)calloc(GENOMEBATCHSZ, sizeof(bool));
  h_guides = (bool*)calloc(GUIDESBATCHSZ, sizeof(bool));
  h_output = (bool*)calloc(OUTPUTBATCHSZ, sizeof(bool));
  
  TOTGUIDES = 136632259;
  exact = (int*) calloc(TOTGUIDES, sizeof(int));

  fexact  = fopen("../Results/exact.txt", "w+");
  fmis1   = fopen("../Results/mis1.txt", "w+");
  fmis2   = fopen("../Results/mis2.txt", "w+");

  fgenome = fopen("chr1.txt", "r"); // GENOME
  while (flaggenome){
    printf("Genome: %d...\n", absidxgen);
    igenome = -1;
    ibingen = 0;
    while ((inp = fgetc(fgenome))!= EOF 
            && igenome < (GENOMEBATCHSZ/NUCBITSZ)){
      if ((char)inp != '\n'){
        bool *base;
        base = getbool((char)inp);
        for (int i = 0; i < NUCBITSZ; i++) h_genome[ibingen + i] = base[i];
        igenome++;
        ibingen += 3;
      }
    }
    if (inp == EOF) flaggenome = false;
    gpuErrchk(hipMemcpy(d_genome, &(h_genome[ngenome]), sizeof(bool)*GENOMEBATCHSZ, hipMemcpyHostToDevice));
    nguides = 0;
    flagguides = true;
    absidxgui = 0; 
    fguides = fopen("guides.txt", "r"); // GUIDES
    while (flagguides){
      printf("Guides: %d...\n", absidxgui);
      iguides = -1;
      ibingui = 0;
      while ((inp = fgetc(fguides)) != EOF && iguides < (GUIDESBATCHSZ/NUCBITSZ)){
        if ((char)inp != '\n'){
          bool * base;
          base = getbool((char)inp);
          for (int  i = 0; i < NUCBITSZ; i++) h_guides[ibingui + i] = base[i];
          iguides++;
          ibingui += 3;
        }
      }
      if (inp == EOF) flagguides = false;
      gpuErrchk(hipMemcpy(d_guides, &(h_guides[nguides]), sizeof(bool)*GUIDESBATCHSZ, hipMemcpyHostToDevice));
      myKernel<<<8, BLOCKSZ>>>(d_genome, d_guides, d_output, GUIDESBATCHSZ);
      gpuErrchk(hipMemcpy(h_output, d_output, sizeof(bool)*OUTPUTBATCHSZ, hipMemcpyDeviceToHost));
      for (int i = 0; i < iguides/GUIDELENGTH; i++) 
        for (int j = 0; j < (igenome - GUIDELENGTH + 1); j++){
          //if (h_output[i*iguides+j]) fprintf(fexact, "%d,%d:", absidxgui + i, absidxgen + j);
          if (h_output[i*iguides+j]) exact[i+absidxgui] += 1;
        }
      nguides += ibingui;
      absidxgui += iguides;
    }
    /*for(int i = 0; i < GUIDESZ; i++)
      fprintf(fexact, "%d:%d,", absidxgui+i, exact[i]);*/
    fclose(fguides);
    ngenome += ibingen;
    absidxgen += igenome;
  }
  fclose(fgenome);
  printf("Number of GUIDES: %d", TOTGUIDES);
  for (int i = 0; i < TOTGUIDES; i++)
    if (exact[i] != 0) fprintf(fexact, "%d:%d,", i, exact[i]);
  fclose(fexact);
  fclose(fmis1);  
  fclose(fmis2);

  free(h_guides);
  free(h_genome);
  free(h_output);
  free(exact);
  hipFree(d_genome);
  hipFree(d_guides);
  hipFree(d_output);
  return 0;
}
